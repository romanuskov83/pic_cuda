#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <hip/driver_types.h>
#include "gpu_code.h"
#include "grid.h"
#define tx threadIdx.x
#define bx blockIdx.x

#define GPU_RESOURCE_UTILIZATION_FACTOR 0.9f

#define EXCHANGE_PARTICLES_BLOCKS (EXCHANGE_PARTICLES_COUNT/PARTICLE_BLOCK_SIZE)


GpuState* gpuInit(int deviceId, int gridNx, int gridNy, int gridNz, Coordinate *gridX, Coordinate *gridY, Coordinate *gridZ) {
    GpuState* result = new GpuState;

    result->deviceId = deviceId;
    result->gridNx = gridNx;
    result->gridNy = gridNy;
    result->gridNz = gridNz;
    result->gridX = gridX;
    result->gridY = gridY;
    result->gridZ = gridZ;

    int gridDim = gridNx*gridNy*gridNz;
    result->gridDim = gridDim;
    int gridPlusOneDim = (gridNx+1)*(gridNy+1)*(gridNz+1);

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(deviceId);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";
        return NULL;
    }

    struct hipDeviceProp_t props;
    cudaStatus = hipGetDeviceProperties(&props, deviceId);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipGetDeviceProperties failed! ";
        return NULL;
    }
    std::cout << "Using " << props.name << " with " << props.multiProcessorCount << " MP\n";

    cudaStatus = hipStreamCreate(&result->modellingStream);
    if (cudaStatus != hipSuccess) {
        std::cerr << "cudaCreateStream failed! modellingStream";
        return NULL;
    }

    cudaStatus = hipStreamCreate(&result->exchangesStream);
    if (cudaStatus != hipSuccess) {
        std::cerr << "cudaCreateStream failed! exchangesStream";
        return NULL;
    }

    result->workingBlocksCount = props.multiProcessorCount*2;
    result->particlesBlocksCount = (int)((props.totalGlobalMem*GPU_RESOURCE_UTILIZATION_FACTOR
                                          -gridDim*3*sizeof(FieldComponent)-gridPlusOneDim*3*sizeof(FieldComponent)
                                          -(gridNx+1+gridNy+1+gridNz+1)*sizeof(Coordinate)
                                          -sizeof(ParticlesBlock)*EXCHANGE_PARTICLES_BLOCKS-gridDim*12*sizeof(FieldComponent)*result->workingBlocksCount)/sizeof(ParticlesBlock));
    std::cout << "Particles blocks count: " << result->particlesBlocksCount << "\n";



    cudaStatus = hipMallocManaged(&result->exchangeCounters, 3*sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocManaged failed! exchangeCounters");
        return NULL;
    }


    cudaStatus = hipMalloc(&result->gridData, (gridNx+gridNy+gridNz+3)*sizeof(Coordinate));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! gridData");
        return NULL;
    }

    cudaStatus = hipMemcpy(result->gridData, gridX, (gridNx+1)*sizeof(Coordinate),hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! gridX");
        return NULL;
    }

    cudaStatus = hipMemcpy(&result->gridData[gridNx+1], gridY, (gridNy+1)*sizeof(Coordinate),hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! gridY");
        return NULL;
    }

    cudaStatus = hipMemcpy(&result->gridData[gridNx+1+gridNy+1], gridZ, (gridNz+1)*sizeof(Coordinate),hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! gridZ");
        return NULL;
    }


    cudaStatus = hipMalloc(&result->particlesBlocks, sizeof(ParticlesBlock)*result->particlesBlocksCount);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! particlesBlocks");
        return NULL;
    }

    cudaStatus = hipMemset(result->particlesBlocks, 0x0, sizeof(ParticlesBlock)*result->particlesBlocksCount);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMemSet failed! particlesBlocks");
        return NULL;
    }

    cudaStatus = hipMalloc(&result->exchangeParticlesBlocks, sizeof(ParticlesBlock)*EXCHANGE_PARTICLES_BLOCKS);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! exchangeParticlesBlocks");
        return NULL;
    }


    result->exchangeParticlesBlocksHost = (ParticlesBlock*)malloc(sizeof(ParticlesBlock)*EXCHANGE_PARTICLES_BLOCKS);
    if (result->exchangeParticlesBlocksHost == NULL) {
        fprintf(stderr, "malloc failed! exchangeParticlesBlocksHost");
        return NULL;

    }


    cudaStatus = hipMalloc(&result->electricData, sizeof(FieldComponent)*((gridNx)*(gridNy+1)*(gridNz+1) + (gridNx+1)*(gridNy)*(gridNz+1) + (gridNx+1)*(gridNy+1)*(gridNz)));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! electricData");
        return NULL;
    }

    cudaStatus = hipMalloc(&result->magneticData, sizeof(FieldComponent)*((gridNx+1)*(gridNy)*(gridNz) + (gridNx)*(gridNy+1)*(gridNz) + (gridNx)*(gridNy)*(gridNz+1)));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! magneticData");
        return NULL;
    }


    result->currentDataHost = (FieldComponent *)malloc(sizeof(FieldComponent)*gridDim*12);

    cudaStatus = hipMalloc(&result->currentData, sizeof(FieldComponent)*gridDim*12*result->workingBlocksCount);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! currentData");
        return NULL;
    }



    return result;
}

bool gpuUpdateFieldsData(GpuState* gpuState,FieldComponent* electricData, FieldComponent* magneticData) {

    hipError_t cudaStatus;

    cudaStatus = hipMemcpy(gpuState->electricData, electricData, sizeof(FieldComponent)*((gpuState->gridNx)*(gpuState->gridNy+1)*(gpuState->gridNz+1) + (gpuState->gridNx+1)*(gpuState->gridNy)*(gpuState->gridNz+1) + (gpuState->gridNx+1)*(gpuState->gridNy+1)*(gpuState->gridNz)),hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! electricData");
        return false;
    }

    cudaStatus = hipMemcpy(gpuState->magneticData, magneticData, sizeof(FieldComponent)*((gpuState->gridNx+1)*(gpuState->gridNy)*(gpuState->gridNz) + (gpuState->gridNx)*(gpuState->gridNy+1)*(gpuState->gridNz) + (gpuState->gridNx)*(gpuState->gridNy)*(gpuState->gridNz+1)),hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! magneticData");
        return false;
    }

    cudaStatus = hipMemset(gpuState->currentData, 0x0, sizeof(FieldComponent)*gpuState->gridNx*gpuState->gridNy*gpuState->gridNz*12);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemset failed! currentData");
        return false;
    }


    return true;
}



/*Exchanges stream*/

__global__ __launch_bounds__(1024, 1) void exchangeParticles(ParticlesBlock* modellingBlocks, int modellingBlocksCount, ParticlesBlock* exchangeBlocks, int* countersGlobal) {
    __shared__ int counters[2];
    if(tx == 0) {
        counters[0] = *countersGlobal;
        counters[1] = 0;
    }

    int totalModelling = modellingBlocksCount*PARTICLE_BLOCK_SIZE;
    int iterations = totalModelling/blockDim.x;

    /* Put new particles in */
    for(int i = 0; i < iterations; i++) {
        int pIdx = i*blockDim.x + tx;
        int bi = pIdx/PARTICLE_BLOCK_SIZE;
        int pi = pIdx%PARTICLE_BLOCK_SIZE;
        int cellIdFlag = modellingBlocks[bi].cellIdFlag[pi];
        if((cellIdFlag & FLAG_MASK) == FLAG_DIED) {
            int old = atomicAdd(counters,-1);
            if(old > 0) {
                int bi2 = (old-1)/PARTICLE_BLOCK_SIZE;
                int pi2 = (old-1)%PARTICLE_BLOCK_SIZE;
                modellingBlocks[bi].rx[pi] = exchangeBlocks[bi2].rx[pi2];
                modellingBlocks[bi].ry[pi] = exchangeBlocks[bi2].ry[pi2];
                modellingBlocks[bi].rz[pi] = exchangeBlocks[bi2].rz[pi2];

                modellingBlocks[bi].px[pi] = exchangeBlocks[bi2].px[pi2];
                modellingBlocks[bi].py[pi] = exchangeBlocks[bi2].py[pi2];
                modellingBlocks[bi].pz[pi] = exchangeBlocks[bi2].pz[pi2];

                modellingBlocks[bi].weight[pi] = exchangeBlocks[bi2].weight[pi2];
                modellingBlocks[bi].currentTime[pi] = exchangeBlocks[bi2].currentTime[pi2];

                modellingBlocks[bi].cellIdFlag[pi] = exchangeBlocks[bi2].cellIdFlag[pi2];
                modellingBlocks[bi].id[pi] = exchangeBlocks[bi2].id[pi2];
            }
        }

    }
    __syncthreads();

    /* not all IN particles were processed - no space - so we take OUT particles starting from Nth position keeping N IN particles for later*/
    if(counters[0] > 0) {
        counters[1] = counters[0];
    }

    __syncthreads();

    /* Put new particles in */
    for(int i = 0; i < iterations; i++) {
        int pIdx = i*blockDim.x + tx;
        int bi = pIdx/PARTICLE_BLOCK_SIZE;
        int pi = pIdx%PARTICLE_BLOCK_SIZE;
        if((modellingBlocks[bi].cellIdFlag[pi] & FLAG_MASK) > FLAG_OK) {
            int old = atomicAdd(counters+1,1);
            if(old < EXCHANGE_PARTICLES_BLOCKS*PARTICLE_BLOCK_SIZE) {
                int bi2 = old/PARTICLE_BLOCK_SIZE;
                int pi2 = old%PARTICLE_BLOCK_SIZE;
                exchangeBlocks[bi2].rx[pi2] = modellingBlocks[bi].rx[pi];
                exchangeBlocks[bi2].ry[pi2] = modellingBlocks[bi].ry[pi];
                exchangeBlocks[bi2].rz[pi2] = modellingBlocks[bi].rz[pi];

                exchangeBlocks[bi2].px[pi2] = modellingBlocks[bi].px[pi];
                exchangeBlocks[bi2].py[pi2] = modellingBlocks[bi].py[pi];
                exchangeBlocks[bi2].pz[pi2] = modellingBlocks[bi].pz[pi];

                exchangeBlocks[bi2].currentTime[pi2] = modellingBlocks[bi].currentTime[pi];
                exchangeBlocks[bi2].weight[pi2] = modellingBlocks[bi].weight[pi];
                exchangeBlocks[bi2].cellIdFlag[pi2] = modellingBlocks[bi].cellIdFlag[pi];
                exchangeBlocks[bi2].id[pi2] = modellingBlocks[bi].id[pi];

                modellingBlocks[bi].cellIdFlag[pi] = FLAG_DIED;
            }
        }
    }
    __syncthreads();
    if(tx == 0) {
        countersGlobal[0] = counters[0];
        countersGlobal[1] = counters[1];
    }
}

bool gpuExchangeParticles(GpuState *state, ParticleInfo *particles, int& countIn, int& countOut) {


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipError_t cudaStatus;

    for(int i = 0; i < countIn; i++) {
        ParticleInfo& info = particles[i];
        int ix = findCell(info.rx,state->gridX, state->gridNx);
        int iy = findCell(info.ry,state->gridY, state->gridNy);
        int iz = findCell(info.rz,state->gridZ, state->gridNz);
        int cellIdx = cel_idx(ix,iy,iz,state->gridNx,state->gridNy,state->gridNz);
        int pi = i%PARTICLE_BLOCK_SIZE;

        ParticlesBlock* pb = &state->exchangeParticlesBlocksHost[i/PARTICLE_BLOCK_SIZE];

        pb->rx[pi] = particles[i].rx;
        pb->ry[pi] = particles[i].ry;
        pb->rz[pi] = particles[i].rz;

        pb->px[pi] = particles[i].px;
        pb->py[pi] = particles[i].py;
        pb->pz[pi] = particles[i].pz;
        pb->id[pi] = particles[i].id;
        pb->currentTime[pi] = particles[i].currentTime;

        pb->weight[pi] = particles[i].weight;

        pb->cellIdFlag[pi] = FLAG_OK | cellIdx;
    }

    state->exchangeCounters[0] = countIn;
    state->exchangeCounters[1] = 0;


    hipEventRecord(start);

    cudaStatus = hipMemcpyAsync(state->exchangeParticlesBlocks,state->exchangeParticlesBlocksHost, sizeof(ParticlesBlock)*EXCHANGE_PARTICLES_BLOCKS,hipMemcpyHostToDevice,state->exchangesStream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! exchangeParticlesBlocks");
        return false;
    }

    cudaStatus = hipStreamSynchronize(state->exchangesStream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! exchangesStream");
        return false;
    }

    dim3 blocks;
    blocks.x = 1;
    blocks.y = 1;
    blocks.z = 1;
    dim3 threads;
    threads.x = 1024;
    threads.y = 1;
    threads.z = 1;
    //std::cout << "exchange particles kernel\n";
    exchangeParticles<<<blocks,threads,0,state->exchangesStream>>>(state->particlesBlocks, state->particlesBlocksCount,state->exchangeParticlesBlocks, state->exchangeCounters);
    cudaStatus = hipStreamSynchronize(state->exchangesStream);
    if(cudaStatus != hipSuccess) {
        std::cerr << "Kernel failed: exchangeParticles " << cudaStatus <<"\n";
        return false;
    }



    printf("AVAILABLE PARTICLES %d PARTICLES TO SEND %d\n",-state->exchangeCounters[0], state->exchangeCounters[1] - (state->exchangeCounters[0] > 0 ? PARTICLE_BLOCK_SIZE*EXCHANGE_PARTICLES_BLOCKS-state->exchangeCounters[0]: 0));


    cudaStatus = hipMemcpyAsync(state->exchangeParticlesBlocksHost,state->exchangeParticlesBlocks, sizeof(ParticlesBlock)*EXCHANGE_PARTICLES_BLOCKS,hipMemcpyDeviceToHost,state->exchangesStream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! exchangeParticlesBlocks");
        return false;
    }

    cudaStatus = hipStreamSynchronize(state->exchangesStream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! exchangesStream");
        return false;
    }


    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "EXCHANGEs DONE IN " << milliseconds << "ms\n";


    for(int i = 0; i < state->exchangeCounters[1]; i++) {
        particles[i].rx = state->exchangeParticlesBlocksHost[i/PARTICLE_BLOCK_SIZE].rx[i%PARTICLE_BLOCK_SIZE];
        particles[i].ry = state->exchangeParticlesBlocksHost[i/PARTICLE_BLOCK_SIZE].ry[i%PARTICLE_BLOCK_SIZE];
        particles[i].rz = state->exchangeParticlesBlocksHost[i/PARTICLE_BLOCK_SIZE].rz[i%PARTICLE_BLOCK_SIZE];

        particles[i].px = state->exchangeParticlesBlocksHost[i/PARTICLE_BLOCK_SIZE].px[i%PARTICLE_BLOCK_SIZE];
        particles[i].py = state->exchangeParticlesBlocksHost[i/PARTICLE_BLOCK_SIZE].py[i%PARTICLE_BLOCK_SIZE];
        particles[i].pz = state->exchangeParticlesBlocksHost[i/PARTICLE_BLOCK_SIZE].pz[i%PARTICLE_BLOCK_SIZE];

        particles[i].weight = state->exchangeParticlesBlocksHost[i/PARTICLE_BLOCK_SIZE].weight[i%PARTICLE_BLOCK_SIZE];
        particles[i].currentTime = state->exchangeParticlesBlocksHost[i/PARTICLE_BLOCK_SIZE].currentTime[i%PARTICLE_BLOCK_SIZE];
        particles[i].cellIdFlag = state->exchangeParticlesBlocksHost[i/PARTICLE_BLOCK_SIZE].cellIdFlag[i%PARTICLE_BLOCK_SIZE];
        particles[i].id = state->exchangeParticlesBlocksHost[i/PARTICLE_BLOCK_SIZE].id[i%PARTICLE_BLOCK_SIZE];

    }

    countIn = state->exchangeCounters[0] > 0 ? state->exchangeCounters[0] : 0;
    countOut = state->exchangeCounters[1];
    return true;
}


/*Modelling stream*/




//__launch_bounds__(PARTICLE_BLOCK_SIZE, 16)

__global__  void makeStep(ParticlesBlock* particlesBlocks, int particlesBlocksCount,const FieldComponent* eX, const FieldComponent* hX, FieldComponent* currentData,const float startTime, const float endTime,const int gridNx, const int gridNy, const int gridNz, const Coordinate* gridDataX, int* exchangeCounters
)
{

    int iterations = (PARTICLE_BLOCK_SIZE*particlesBlocksCount)/(blockDim.x*gridDim.x);
    int i;
    int pIdx;
    int bi=0;
    int pi=0;

    Coordinate rx;
    Coordinate ry;
    Coordinate rz;
    Coordinate px;
    Coordinate py;
    Coordinate pz;
    float currentTime=0;

    for(i = 0; i < iterations; ++i) {
        pIdx = blockDim.x*gridDim.x*i + blockDim.x*bx+tx;
        bi = pIdx/PARTICLE_BLOCK_SIZE;
        pi = pIdx%PARTICLE_BLOCK_SIZE;
        int flag = particlesBlocks[bi].cellIdFlag[pi];
        if((flag & FLAG_MASK) == FLAG_OK) {
            currentTime = particlesBlocks[bi].currentTime[pi];
            if(currentTime < endTime && currentTime >= startTime)
                break;
        }
    }


    __syncthreads();

    if(i == iterations) {
        return;
    }


    exchangeCounters[2] = 1;

    const int cellId = particlesBlocks[bi].cellIdFlag[pi] & CELL_ID_MASK;
    const int ix = x_idx(cellId,gridNx,gridNy,gridNz);
    const int iy = y_idx(cellId,gridNx,gridNy,gridNz);
    const int iz = z_idx(cellId,gridNx,gridNy,gridNz);

    const Coordinate* gridDataY = gridDataX+gridNx+1;
    const Coordinate* gridDataZ = gridDataX+gridNx+gridNy+2;

    const FieldComponent* eY = eX+gridNx*(gridNy+1)*(gridNz+1);
    const FieldComponent* eZ = eX+gridNx*(gridNy+1)*(gridNz+1)+(gridNx+1)*gridNy*(gridNz+1);

    const FieldComponent* hY = hX+(gridNx+1)*gridNy*gridNz;
    const FieldComponent* hZ = hX+(gridNx+1)*gridNy*gridNz+gridNx*(gridNy+1)*gridNz;


    const Coordinate fromX = gridDataX[ix];
    const Coordinate toX = gridDataX[ix+1];
    const Coordinate fromY = gridDataY[iy];
    const Coordinate toY = gridDataY[iy+1];
    const Coordinate fromZ = gridDataZ[iz];
    const Coordinate toZ = gridDataZ[iz+1];

    rx = particlesBlocks[bi].rx[pi];
    ry = particlesBlocks[bi].ry[pi];
    rz = particlesBlocks[bi].rz[pi];

    px = particlesBlocks[bi].px[pi];
    py = particlesBlocks[bi].py[pi];
    pz = particlesBlocks[bi].pz[pi];

    Coordinate  u02= px*px + py*py + pz*pz;
    Coordinate  q_v = C/sqrtf(1.0f+u02);
    Coordinate vx = px*q_v;
    Coordinate vy = py*q_v;
    Coordinate vz = pz*q_v;
//    Coordinate v = norm3df(vx,vy,vz);
    


    Coordinate toBorderTimeX;
    if(vx > 0) {
        toBorderTimeX = (toX-rx)/vx;
    } else if(vx < 0) {
        toBorderTimeX = (fromX-rx)/vx;
    } else {
        toBorderTimeX = 1e9;
    }


    Coordinate toBorderTimeY;
    if(vy > 0) {
        toBorderTimeY = (toY-ry)/vy;
    } else if(vy < 0) {
        toBorderTimeY = (fromY-ry)/vy;
    } else {
        toBorderTimeY = 1e9;
    }

    Coordinate toBorderTimeZ;
    if(vz > 0) {
        toBorderTimeZ = (toZ-rz)/vz;
    } else if(vz < 0) {
        toBorderTimeZ = (fromZ-rz)/vz;
    } else {
        toBorderTimeZ = 1e9;
    }

    Coordinate toBorderTime = min(toBorderTimeX,min(toBorderTimeY,toBorderTimeZ));
    const Coordinate time = min(toBorderTime,endTime-currentTime);

    //currents_1(Me*Qe*particlesBlocks[bi].weight[pi],ix,iy,iz,rx,ry,rz,vx*time,vy*time,vz*time);

    //!коэффициенты для целых точек
    float cx =(rx-fromX)/(toX-fromX);
    float cy =(ry-fromY)/(toY-fromY);
    float cz =(rz-fromZ)/(toZ-fromZ);

    int ixc;
    int iyc=0;
    int izc=0;
    float cxc;
    float cyc=0;
    float czc=0;

    if(rx < (fromX+toX)/2) {
        if(ix == 0) {
            ixc = ix;
            cxc = 0.0f;
        } else {
            ixc = ix-1;
            cxc = (rx - (gridDataX[ixc]+fromX)/2)/((fromX+toX)/2-(gridDataX[ixc]+fromX)/2);
        }
    } else {
        if(ix == gridNx-1) {
            ixc = ix-1;
            cxc = 1.0f;
        } else {
            ixc = ix;
            cxc = (rx - (fromX+toX)/2)/((gridDataX[ixc+2]+toX)/2-(fromX+toX)/2);
        }
    };


    if(ry < (fromY+toY)/2) {
        if(iy == 0) {
            iyc = iy;
            cyc = 0.0f;
        } else {
            iyc = iy-1;
            cyc = (ry - (gridDataY[iyc]+fromY)/2)/((fromY+toY)/2-(gridDataY[iyc]+fromY)/2);
        }
    } else {
        if(iy == gridNy-1) {
            iyc = iy-1;
            cyc = 1.0f;
        } else {
            iyc = iy;
            cyc = (ry - (fromY+toY)/2)/((gridDataY[iyc+2]+toY)/2-(fromY+toY)/2);
        }
    };

    if(rz < (fromZ+toZ)/2) {
        if(iz == 0) {
            izc = iz;
            czc = 0.0f;
        } else {
            izc = iz-1;
            czc = (rz - (gridDataZ[izc]+fromZ)/2)/((fromZ+toZ)/2-(gridDataZ[izc]+fromZ)/2);
        }
    } else {
        if(iz == gridNz-1) {
            izc = iz-1;
            czc = 1.0f;
        } else {
            izc = iz;
            czc = (rz - (fromZ+toZ)/2)/((gridDataZ[izc+2]+toZ)/2-(fromZ+toZ)/2);
        }
    };


//    printf("ZZZZ %f\n",(fromX+toX)/2);
//    printf("ZZZZ %f\n",(gridDataX[ix+2]+toX)/2);
//    printf("\t\t%f %f %f - %d %d %d / %f %f %f - %d %d %d / %f %f %f\n", rx,ry,rz,ix,iy,iz,cx,cy,cz,ixc,iyc,izc,cxc,cyc,czc);

    if(toBorderTime > endTime-currentTime) {
        rx += vx*time;
        ry += vy*time;
        rz += vz*time;
        currentTime = endTime;

    } else {
        //printf("REACHED BORDER\n");

        rx += vx*toBorderTime;
        ry += vy*toBorderTime;
        rz += vz*toBorderTime;
        currentTime += toBorderTime;

        if(toBorderTime == toBorderTimeX) {
            if(px > 0) {
                rx = toX;
                if(ix == gridNx-1) {
                    particlesBlocks[bi].cellIdFlag[pi] = (particlesBlocks[bi].cellIdFlag[pi] & CELL_ID_MASK) | FLAG_FLEW_PLUS_X;
                } else {
                    particlesBlocks[bi].cellIdFlag[pi] = (particlesBlocks[bi].cellIdFlag[pi] & FLAG_MASK) | (cellId+gridNz*gridNy);
                }
            } else {
                rx = fromX;
                if(ix == 0) {
                    particlesBlocks[bi].cellIdFlag[pi] = (particlesBlocks[bi].cellIdFlag[pi] & CELL_ID_MASK) | FLAG_FLEW_MINUS_X;
                } else {
                    particlesBlocks[bi].cellIdFlag[pi] = (particlesBlocks[bi].cellIdFlag[pi] & FLAG_MASK) | (cellId-gridNz*gridNy);
                }
            }

        } else if(toBorderTime == toBorderTimeY) {
            if(py > 0) {
                ry = toY;
                if(iy == gridNy-1) {
                    particlesBlocks[bi].cellIdFlag[pi] = (particlesBlocks[bi].cellIdFlag[pi] & CELL_ID_MASK) | FLAG_FLEW_PLUS_Y;
                } else {
                    particlesBlocks[bi].cellIdFlag[pi] = (particlesBlocks[bi].cellIdFlag[pi] & FLAG_MASK) | (cellId+gridNz);
                }
            } else {
                ry = fromY;
                if(iy == 0) {
                    particlesBlocks[bi].cellIdFlag[pi] = (particlesBlocks[bi].cellIdFlag[pi] & CELL_ID_MASK) | FLAG_FLEW_MINUS_Y;
                } else {
                    particlesBlocks[bi].cellIdFlag[pi] = (particlesBlocks[bi].cellIdFlag[pi] & FLAG_MASK) | (cellId-gridNz);
                }
            }
        } else {
            if(pz > 0) {
                rz = toZ;
                if(iz == gridNz-1) {
                    particlesBlocks[bi].cellIdFlag[pi] = (particlesBlocks[bi].cellIdFlag[pi] & CELL_ID_MASK) | FLAG_FLEW_PLUS_Z;
                } else {
                    particlesBlocks[bi].cellIdFlag[pi] = (particlesBlocks[bi].cellIdFlag[pi] & FLAG_MASK) | (cellId+1);
                }
            } else {
                rz = fromZ;
                if(iz == 0) {
                    particlesBlocks[bi].cellIdFlag[pi] = (particlesBlocks[bi].cellIdFlag[pi] & CELL_ID_MASK) | FLAG_FLEW_MINUS_Z;
                } else {
                    particlesBlocks[bi].cellIdFlag[pi] = (particlesBlocks[bi].cellIdFlag[pi] & FLAG_MASK) | (cellId-1);
                }
            }
        }

    }


    const float cfield = -Qe*time*Ee/(C*Me);


    const float yex=cfield*(eX[cel_idx(ixc,iy,iz,gridNx,gridNy+1,gridNz+1)]*(1.0f-cxc)*(1.0f-cy)*(1.0f-cz) +
                      eX[cel_idx(ixc+1,iy,iz,gridNx,gridNy+1,gridNz+1)]*(cxc)*(1.0f-cy)*(1.0f-cz) +
                      eX[cel_idx(ixc,iy+1,iz,gridNx,gridNy+1,gridNz+1)]*(1.0f-cxc)*(cy)*(1.0f-cz) +
                      eX[cel_idx(ixc+1,iy+1,iz,gridNx,gridNy+1,gridNz+1)]*(cxc)*(cy)*(1.0f-cz) +
                      eX[cel_idx(ixc,iy,iz+1,gridNx,gridNy+1,gridNz+1)]*(1.0f-cxc)*(1.0f-cy)*(cz) +
                      eX[cel_idx(ixc+1,iy,iz+1,gridNx,gridNy+1,gridNz+1)]*(cxc)*(1.0f-cy)*(cz) +
                      eX[cel_idx(ixc,iy+1,iz+1,gridNx,gridNy+1,gridNz+1)]*(1.0f-cxc)*(cy)*(cz) +
                      eX[cel_idx(ixc+1,iy+1,iz+1,gridNx,gridNy+1,gridNz+1)]*(cxc)*(cy)*(cz));


    const float yey=cfield*(eY[cel_idx(ix,iyc,iz,gridNx+1,gridNy,gridNz+1)]*(1.0f-cx)*(1.0f-cyc)*(1.0f-cz) +
                      eY[cel_idx(ix+1,iyc,iz,gridNx+1,gridNy,gridNz+1)]*(cx)*(1.0f-cyc)*(1.0f-cz) +
                      eY[cel_idx(ix,iyc+1,iz,gridNx+1,gridNy,gridNz+1)]*(1.0f-cx)*(cyc)*(1.0f-cz) +
                      eY[cel_idx(ix+1,iyc+1,iz,gridNx+1,gridNy,gridNz+1)]*(cx)*(cyc)*(1.0f-cz) +
                      eY[cel_idx(ix,iyc,iz+1,gridNx+1,gridNy,gridNz+1)]*(1.0f-cx)*(1.0f-cyc)*(cz) +
                      eY[cel_idx(ix+1,iyc,iz+1,gridNx+1,gridNy,gridNz+1)]*(cx)*(1.0f-cyc)*(cz) +
                      eY[cel_idx(ix,iyc+1,iz+1,gridNx+1,gridNy,gridNz+1)]*(1.0f-cx)*(cyc)*(cz) +
                      eY[cel_idx(ix+1,iyc+1,iz+1,gridNx+1,gridNy,gridNz+1)]*(cx)*(cyc)*(cz));

    const float yez=cfield*(eZ[cel_idx(ix,iy,izc,gridNx+1,gridNy+1,gridNz)]*(1.0f-cx)*(1.0f-cy)*(1.0f-czc) +
                      eZ[cel_idx(ix+1,iy,izc,gridNx+1,gridNy+1,gridNz)]*(cx)*(1.0f-cy)*(1.0f-czc) +
                      eZ[cel_idx(ix,iy+1,izc,gridNx+1,gridNy+1,gridNz)]*(1.0f-cx)*(cy)*(1.0f-czc) +
                      eZ[cel_idx(ix+1,iy+1,izc,gridNx+1,gridNy+1,gridNz)]*(cx)*(cy)*(1.0f-czc) +
                      eZ[cel_idx(ix,iy,izc+1,gridNx+1,gridNy+1,gridNz)]*(1.0f-cx)*(1.0f-cy)*(czc) +
                      eZ[cel_idx(ix+1,iy,izc+1,gridNx+1,gridNy+1,gridNz)]*(cx)*(1.0f-cy)*(czc) +
                      eZ[cel_idx(ix,iy+1,izc+1,gridNx+1,gridNy+1,gridNz)]*(1.0f-cx)*(cy)*(czc) +
                      eZ[cel_idx(ix+1,iy+1,izc+1,gridNx+1,gridNy+1,gridNz)]*(cx)*(cy)*(czc));

    float yhx=cfield*(hX[cel_idx(ix,iyc,izc,gridNx+1,gridNy,gridNz)]*(1.0f-cx)*(1.0f-cyc)*(1.0f-czc)+
                      hX[cel_idx(ix+1,iyc,izc,gridNx+1,gridNy,gridNz)]*(cx)*(1.0f-cyc)*(1.0f-czc)+
                      hX[cel_idx(ix,iyc+1,izc,gridNx+1,gridNy,gridNz)]*(1.0f-cx)*(cyc)*(1.0f-czc)+
                      hX[cel_idx(ix+1,iyc+1,izc,gridNx+1,gridNy,gridNz)]*(cx)*(cyc)*(1.0f-czc)+
                      hX[cel_idx(ix,iyc,izc+1,gridNx+1,gridNy,gridNz)]*(1.0f-cx)*(1.0f-cyc)*(czc)+
                      hX[cel_idx(ix+1,iyc,izc+1,gridNx+1,gridNy,gridNz)]*(cx)*(1.0f-cyc)*(czc)+
                      hX[cel_idx(ix,iyc+1,izc+1,gridNx+1,gridNy,gridNz)]*(1.0f-cx)*(cyc)*(czc)+
                      hX[cel_idx(ix+1,iyc+1,izc+1,gridNx+1,gridNy,gridNz)]*(cx)*(cyc)*(czc));

    float yhy=cfield*(hY[cel_idx(ixc,iy,izc,gridNx,gridNy+1,gridNz)]*(1.0f-cxc)*(1.0f-cy)*(1.0f-czc)+
                      hY[cel_idx(ixc+1,iy,izc,gridNx,gridNy+1,gridNz)]*(cxc)*(1.0f-cy)*(1.0f-czc)+
                      hY[cel_idx(ixc,iy+1,izc,gridNx,gridNy+1,gridNz)]*(1.0f-cxc)*(cy)*(1.0f-czc)+
                      hY[cel_idx(ixc+1,iy+1,izc,gridNx,gridNy+1,gridNz)]*(cxc)*(cy)*(1.0f-czc)+
                      hY[cel_idx(ixc,iy,izc+1,gridNx,gridNy+1,gridNz)]*(1.0f-cxc)*(1.0f-cy)*(czc)+
                      hY[cel_idx(ixc+1,iy,izc+1,gridNx,gridNy+1,gridNz)]*(cxc)*(1.0f-cy)*(czc)+
                      hY[cel_idx(ixc,iy+1,izc+1,gridNx,gridNy+1,gridNz)]*(1.0f-cxc)*(cy)*(czc)+
                      hY[cel_idx(ixc+1,iy+1,izc+1,gridNx,gridNy+1,gridNz)]*(cxc)*(cy)*(czc));
    
    float yhz=cfield*(hZ[cel_idx(ixc,iyc,iz,gridNx,gridNy,gridNz+1)]*(1.0f-cxc)*(1.0f-cyc)*(1.0f-cz)+
                      hZ[cel_idx(ixc+1,iyc,iz,gridNx,gridNy,gridNz+1)]*(cxc)*(1.0f-cyc)*(1.0f-cz)+
                      hZ[cel_idx(ixc,iyc+1,iz,gridNx,gridNy,gridNz+1)]*(1.0f-cxc)*(cyc)*(1.0f-cz)+
                      hZ[cel_idx(ixc+1,iyc+1,iz,gridNx,gridNy,gridNz+1)]*(cxc)*(cyc)*(1.0f-cz)+
                      hZ[cel_idx(ixc,iyc,iz+1,gridNx,gridNy,gridNz+1)]*(1.0f-cxc)*(1.0f-cyc)*(cz)+
                      hZ[cel_idx(ixc+1,iyc,iz+1,gridNx,gridNy,gridNz+1)]*(cxc)*(1.0f-cyc)*(cz)+
                      hZ[cel_idx(ixc,iyc+1,iz+1,gridNx,gridNy,gridNz+1)]*(1.0f-cxc)*(cyc)*(cz)+
                      hZ[cel_idx(ixc+1,iyc+1,iz+1,gridNx,gridNy,gridNz+1)]*(cxc)*(cyc)*(cz));

    printf("%f %f %f\n",rx,ry,rz);

    //!ux,uy,uz calculation
    px-=yex;
    py-=yey;
    pz-=yez;

    const double d2gm=0.5/sqrtf(1.0f + 0.25f*sqr(2*px+yex) + 0.25f*sqr(2*py+yey) + 0.25f*sqr(2*pz+yez));


    const double _bx=yhx*d2gm;
    const double _by=yhy*d2gm;
    const double _bz=yhz*d2gm;

    const double fx=px-(yex+(yhz*py-yhy*pz)*d2gm);
    const double fy=py-(yey+(yhx*pz-yhz*px)*d2gm);
    const double fz=pz-(yez+(yhy*px-yhx*py)*d2gm);
    const double bx2=_bx*_bx;
    const double by2=_by*_by;
    const double bz2=_bz*_bz;
    const double bxy=_bx*_by;
    const double bxz=_bx*_bz;
    const double byz=_by*_bz;

    const double ddt=1.0f/(1.0f+bx2+by2+bz2);
    //printf("AAA %le\n",ddt);

    px=ddt*(fx*(1.0+bx2)+fy*(bxy-_bz)  +fz*(bxz+_by));
    py=ddt*(fx*(bxy+_bz)  +fy*(1.0+by2)+fz*(byz-_bx));
    pz=ddt*(fx*(bxz-_by)  +fy*(byz+_bx)  +fz*(1.0+bz2));


    particlesBlocks[bi].rx[pi] = rx;
    particlesBlocks[bi].ry[pi] = ry;
    particlesBlocks[bi].rz[pi] = rz;

    particlesBlocks[bi].px[pi] = px;
    particlesBlocks[bi].py[pi] = py;
    particlesBlocks[bi].pz[pi] = pz;

    particlesBlocks[bi].currentTime[pi] = currentTime;


    const int offset = 12*(bx*gridNx*gridNy*gridNz+cellId);

    atomicAdd(currentData+offset,px);
    atomicAdd(currentData+offset+1,py);
    atomicAdd(currentData+offset+2,pz);

    atomicAdd(currentData+offset+3,px);
    atomicAdd(currentData+offset+4,py);
    atomicAdd(currentData+offset+5,pz);

    atomicAdd(currentData+offset+6,px);
    atomicAdd(currentData+offset+7,py);
    atomicAdd(currentData+offset+8,pz);

    atomicAdd(currentData+offset+9,px);
    atomicAdd(currentData+offset+10,py);
    atomicAdd(currentData+offset+11,pz);

}

__global__  void collectCurrentData(FieldComponent* currentData, int dataRes, int copiesCount)
{
    int i = bx*blockDim.x+tx;
    if(i >= dataRes)
        return;
    for(int j = 1; j < copiesCount;j++)
    {
        currentData[i] += currentData[j*dataRes+i];
    }
}

bool gpuMakeStep(GpuState* state, float startTime, float endTime) {

    hipEvent_t start, stop,startT, stopT;
    hipEventCreate(&start);
    hipEventCreate(&startT);
    hipEventCreate(&stop);
    hipEventCreate(&stopT);

    hipError_t cudaStatus;
    dim3 blocks;
    dim3 threads;

    hipEventRecord(startT);



    hipMemsetAsync(state->currentData, 0x0, sizeof(FieldComponent)*state->gridDim*12*state->workingBlocksCount,state->modellingStream);
    cudaStatus = hipStreamSynchronize(state->modellingStream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemsetAsync failed! currentData");
        return NULL;
    }


    int x=0;
    float milliseconds = 0;
    do {
        x++;
        state->exchangeCounters[2] = 0;




        blocks.x = state->workingBlocksCount;
        blocks.y = 1;
        blocks.z = 1;
        threads.x = 1024;
        threads.y = 1;
        threads.z = 1;

        //hipEventRecord(start);
        makeStep << < blocks, threads, 0, state->modellingStream >> >
                                          (state->particlesBlocks, state->particlesBlocksCount, state->electricData, state->magneticData, state->currentData, startTime, endTime, state->gridNx, state->gridNy, state->gridNz, state->gridData, state->exchangeCounters
                                          );
        cudaStatus = hipStreamSynchronize(state->modellingStream);
        if (cudaStatus != hipSuccess) {
            std::cerr << "Kernel failed: makeStep " << cudaStatus << "\n";
            return false;
        }

        //hipEventRecord(stop);
        //hipEventSynchronize(stop);
        //milliseconds = 0;
        //hipEventElapsedTime(&milliseconds, start, stop);
        //std::cout << "MODELLING DONE IN " << milliseconds << "ms " << x << "iteration\n";

    } while (state->exchangeCounters[2] != 0);

    hipEventRecord(stopT);
    hipEventSynchronize(stopT);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startT, stopT);
    //std::cout << "TOTAL " << milliseconds << "ms " << x << "iterations\n";

    int blocksCount = state->gridDim*12/1024;
    if(state->gridDim*12%1024 != 0) {
        ++blocksCount;
    }

    blocks.x = blocksCount;
    blocks.y = 1;
    blocks.z = 1;
    threads.x = 1024;
    threads.y = 1;
    threads.z = 1;
    collectCurrentData<< < blocks, threads, 0, state->modellingStream >>>(state->currentData, 12*state->gridDim,state->workingBlocksCount);
    cudaStatus = hipStreamSynchronize(state->modellingStream);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Kernel failed: makeStep " << cudaStatus << "\n";
        return false;
    }

    hipMemcpyAsync(state->currentDataHost,state->currentData, sizeof(FieldComponent)*state->gridDim*12,hipMemcpyDeviceToHost,state->modellingStream);
    cudaStatus = hipStreamSynchronize(state->modellingStream);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpyAsync failed: currentData " << cudaStatus << "\n";
        return false;
    }


    return true;
}

